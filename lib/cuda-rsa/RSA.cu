#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <iostream>
#include <fstream>
#include <string.h>
#include <math.h>
#include "RSA.h"

__device__
long long int mod(int base, int exponent, int den)
{
    long long int ret = 1;

    for (int i = 0; i < exponent; i++)
    {
        ret *= base;
        ret %= den;
    }

    return ret;
}

__global__
void rsa(int* num, int* key, int* den, int* result)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int temp;

    temp = mod(num[i], *key, *den);
    atomicExch(&result[i], temp);
}


// #define BS 100

// int p, q, n, t, numChars, tpb = 1024;
// int bpg;
// int e[BS], d[BS], temp[BS], j, m[BS],
//     en[BS], mm[BS], res[BS], i;

// float time_enc_gpu, time_dec_gpu = 0.0;

// char msg[BS];

int p, q, n, t, numChars, tpb = 1024;
int bpg;
int* e, *d, *temp, j, *m, *en, *mm, *res, i;

float time_enc_gpu, time_dec_gpu = 0.0;

char* msg;

int getFileSize(const std::string& filename) {
    std::ifstream file(filename, std::ifstream::ate | std::ifstream::binary);
    if (!file.is_open()) {
        return -1; // Return -1 if there was an error opening the file
    }

    return static_cast<int>(file.tellg());
}


int main(int argc, char* argv[]) {
    p = 157;
    q = 373;

    std::string filePath = argv[1];
    int fileSize = getFileSize(filePath);
    if (fileSize == -1) {
        std::cout << "Error opening the file." << std::endl;
        return 1;
    }

    int BS = fileSize;

    e = new int[BS];
    d = new int[BS];
    temp = new int[BS];
    j = 0;
    m = new int[BS];
    en = new int[BS];
    mm = new int[BS];
    res = new int[BS];
    i = 0;

    msg = new char[BS];

    srand((unsigned) time(NULL));

    msg = new char[BS + 1]; // Allocate space for BS characters plus the null terminator

    std::ifstream inputFile(filePath);
    if (!inputFile.is_open()) {
        std::cout << "Error opening the file." << std::endl;
        delete[] msg; // Clean up dynamically allocated memory
        return 1;
    }

    inputFile.read(msg, BS); // Read BS characters from the file

    numChars = strlen(msg) - 1;
    msg[numChars] = '\0'; // Add null terminator to end of string
    inputFile.close();

    bpg = (numChars + tpb - 1) / tpb;

    for (i = 0; msg[i] != '\0'; i++) {
        m[i] = msg[i];
        mm[i] = msg[i] - 96;
    }

    n = p * q;
    t = (p - 1) * (q - 1);
    fc();

    enc_gpu();
    dec_gpu();
    return 0;
}

void fc() {
    int k = 0;
    long int kv = 1;
    for (int i = 2; i < t && k < 99; i++) {
        if (t % i == 0)
            continue;
        bool ip = true;
        int j = sqrt(i);
        for (int mn = 2; mn <= j; mn++) {
            if (i % mn == 0) {
                ip = false;
                break;
            }
        }
        if (ip && i != p && i != q) {
            e[k] = i;
            long int x = e[k];
            while (true) {
                kv = kv + t;
                if (kv % x == 0) {
                    d[k] = kv / x;
                    k++;
                    break;
                }
            }
        }
    }
}
void enc_gpu() {
    // Allocate and copy memory on the GPU
    hipSetDevice(1);
    
    int key = e[0];
    int *dev_num, *dev_key, *dev_den, *dev_res;
    
    hipMalloc((void**)&dev_num, numChars * sizeof(int));
    hipMalloc((void**)&dev_key, sizeof(int));
    hipMalloc((void**)&dev_den, sizeof(int));
    hipMalloc((void**)&dev_res, numChars * sizeof(int));
    
    hipMemcpy(dev_num, mm, numChars * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_key, &key, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_den, &n, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_res, res, numChars * sizeof(int), hipMemcpyHostToDevice);
    
    // Encryption on the GPU
    hipEvent_t start_encrypt, stop_encrypt;
    hipEventCreate(&start_encrypt);
    hipEventCreate(&stop_encrypt);
    hipEventRecord(start_encrypt);
    
    rsa<<<bpg, tpb>>>(dev_num, dev_key, dev_den, dev_res);
    
    hipEventRecord(stop_encrypt);
    hipEventSynchronize(stop_encrypt);
    hipDeviceSynchronize();
    
    // Calculate elapsed time
    float time_enc_gpu = 0;
    hipEventElapsedTime(&time_enc_gpu, start_encrypt, stop_encrypt);
    time_enc_gpu /= 1000;
    
    // Copy the result back to host and free GPU memory
    hipMemcpy(res, dev_res, numChars * sizeof(int), hipMemcpyDeviceToHost);
    
    hipFree(dev_num);
    hipFree(dev_key);
    hipFree(dev_den);
    hipFree(dev_res);
    
    // Write encrypted result to a file
    FILE *fp = fopen("encrypted_gpu.txt", "wb");
    if (fp != NULL) {
        for (int i = 0; i < numChars; i++) {
            fprintf(fp, "%d", res[i] + 96);
        }
        fclose(fp);
    }
}

void dec_gpu() {
    // Allocate and copy memory on the GPU
    hipSetDevice(1);
    
    int key = d[0];
    int *dev_num, *dev_key, *dev_den, *dev_res;
    
    hipMalloc((void**)&dev_num, numChars * sizeof(int));
    hipMalloc((void**)&dev_key, sizeof(int));
    hipMalloc((void**)&dev_den, sizeof(int));
    hipMalloc((void**)&dev_res, numChars * sizeof(int));
    
    hipMemcpy(dev_num, res, numChars * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_key, &key, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_den, &n, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_res, res, numChars * sizeof(int), hipMemcpyHostToDevice);
    
    // Decryption on the GPU
    hipEvent_t start_decrypt, stop_decrypt;
    hipEventCreate(&start_decrypt);
    hipEventCreate(&stop_decrypt);
    hipEventRecord(start_decrypt);
    
    printf("GPU starts decrypting...\n");
    rsa<<<bpg, tpb>>>(dev_num, dev_key, dev_den, dev_res);
    
    hipEventRecord(stop_decrypt);
    hipEventSynchronize(stop_decrypt);
    hipDeviceSynchronize();
    
    // Calculate elapsed time
    float time_dec_gpu = 0;
    hipEventElapsedTime(&time_dec_gpu, start_decrypt, stop_decrypt);
    time_dec_gpu /= 1000;
    
    // Copy the result back to host and free GPU memory
    hipMemcpy(res, dev_res, numChars * sizeof(int), hipMemcpyDeviceToHost);
    
    hipFree(dev_num);
    hipFree(dev_key);
    hipFree(dev_den);
    hipFree(dev_res);
    
    // Write decrypted result to a file
    FILE *fp = fopen("decrypted_gpu.txt", "wb");
    if (fp != NULL) {
        for (int i = 0; i < numChars; i++) {
            fprintf(fp, "%c", res[i] + 96);
        }
        fprintf(fp, "\n");
        fclose(fp);
    }
}
