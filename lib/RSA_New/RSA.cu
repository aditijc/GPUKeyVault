#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>
#include <math.h>
#include "RSA.h"

__device__
long long int mod(int base, int exponent, int den)
{
    long long int ret = 1;

    for (int i = 0; i < exponent; i++)
    {
        ret *= base;
        ret %= den;
    }

    return ret;
}

__global__
void rsa(int* num, int* key, int* den, int* result)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int temp;

    temp = mod(num[i], *key, *den);
    atomicExch(&result[i], temp);
}


#define BS 100

int p, q, n, t, numChars, tpb = 1024;
int bpg;
int e[BS], d[BS], temp[BS], j, m[BS],
    en[BS], mm[BS], res[BS], i;

float time_enc_gpu, time_dec_gpu = 0.0;

char msg[BS];



int main() {
    p = 157;
    q = 373;

    srand((unsigned) time(NULL));

    FILE *fp = fopen("input.txt", "wb");
    if (fp != NULL) {
        for (int k = 0; k < BS; k++) {
            int r = rand() % 26;
            fprintf(fp, "%c", r + 97);
        }
        fprintf(fp, "\n");
        fclose(fp);
    }

    FILE *f = fopen("input.txt", "r");
    if (f == NULL) {
        perror("Error opening file");
        return 1;
    }

    if (fgets(msg, BS, f) != NULL) {
        printf("Reading input file...");
    }
    fclose(f);
    numChars = strlen(msg) - 1;
    msg[numChars] = '\0';

    bpg = (numChars + tpb - 1) / tpb;

    for (i = 0; msg[i] != '\0'; i++) {
        m[i] = msg[i];
        mm[i] = msg[i] - 96;
    }

    n = p * q;
    t = (p - 1) * (q - 1);
    fc();

    enc_gpu();
    dec_gpu();
    return 0;
}

void fc() {
    int k = 0;
    long int kv = 1;
    for (int i = 2; i < t && k < 99; i++) {
        if (t % i == 0)
            continue;
        bool ip = true;
        int j = sqrt(i);
        for (int mn = 2; mn <= j; mn++) {
            if (i % mn == 0) {
                ip = false;
                break;
            }
        }
        if (ip && i != p && i != q) {
            e[k] = i;
            long int x = e[k];
            while (true) {
                kv = kv + t;
                if (kv % x == 0) {
                    d[k] = kv / x;
                    k++;
                    break;
                }
            }
        }
    }
}
void enc_gpu() {
    // Allocate and copy memory on the GPU
    hipSetDevice(1);
    
    int key = e[0];
    int *dev_num, *dev_key, *dev_den, *dev_res;
    
    hipMalloc((void**)&dev_num, numChars * sizeof(int));
    hipMalloc((void**)&dev_key, sizeof(int));
    hipMalloc((void**)&dev_den, sizeof(int));
    hipMalloc((void**)&dev_res, numChars * sizeof(int));
    
    hipMemcpy(dev_num, mm, numChars * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_key, &key, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_den, &n, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_res, res, numChars * sizeof(int), hipMemcpyHostToDevice);
    
    // Encryption on the GPU
    hipEvent_t start_encrypt, stop_encrypt;
    hipEventCreate(&start_encrypt);
    hipEventCreate(&stop_encrypt);
    hipEventRecord(start_encrypt);
    
    rsa<<<bpg, tpb>>>(dev_num, dev_key, dev_den, dev_res);
    
    hipEventRecord(stop_encrypt);
    hipEventSynchronize(stop_encrypt);
    hipDeviceSynchronize();
    
    // Calculate elapsed time
    float time_enc_gpu = 0;
    hipEventElapsedTime(&time_enc_gpu, start_encrypt, stop_encrypt);
    time_enc_gpu /= 1000;
    
    // Copy the result back to host and free GPU memory
    hipMemcpy(res, dev_res, numChars * sizeof(int), hipMemcpyDeviceToHost);
    
    hipFree(dev_num);
    hipFree(dev_key);
    hipFree(dev_den);
    hipFree(dev_res);
    
    // Write encrypted result to a file
    FILE *fp = fopen("encrypted_gpu.txt", "wb");
    if (fp != NULL) {
        for (int i = 0; i < numChars; i++) {
            fprintf(fp, "%d", res[i] + 96);
        }
        fclose(fp);
    }
}

void dec_gpu() {
    // Allocate and copy memory on the GPU
    hipSetDevice(1);
    
    int key = d[0];
    int *dev_num, *dev_key, *dev_den, *dev_res;
    
    hipMalloc((void**)&dev_num, numChars * sizeof(int));
    hipMalloc((void**)&dev_key, sizeof(int));
    hipMalloc((void**)&dev_den, sizeof(int));
    hipMalloc((void**)&dev_res, numChars * sizeof(int));
    
    hipMemcpy(dev_num, res, numChars * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_key, &key, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_den, &n, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_res, res, numChars * sizeof(int), hipMemcpyHostToDevice);
    
    // Decryption on the GPU
    hipEvent_t start_decrypt, stop_decrypt;
    hipEventCreate(&start_decrypt);
    hipEventCreate(&stop_decrypt);
    hipEventRecord(start_decrypt);
    
    printf("GPU starts decrypting...\n");
    rsa<<<bpg, tpb>>>(dev_num, dev_key, dev_den, dev_res);
    
    hipEventRecord(stop_decrypt);
    hipEventSynchronize(stop_decrypt);
    hipDeviceSynchronize();
    
    // Calculate elapsed time
    float time_dec_gpu = 0;
    hipEventElapsedTime(&time_dec_gpu, start_decrypt, stop_decrypt);
    time_dec_gpu /= 1000;
    
    // Copy the result back to host and free GPU memory
    hipMemcpy(res, dev_res, numChars * sizeof(int), hipMemcpyDeviceToHost);
    
    hipFree(dev_num);
    hipFree(dev_key);
    hipFree(dev_den);
    hipFree(dev_res);
    
    // Write decrypted result to a file
    FILE *fp = fopen("decrypted_gpu.txt", "wb");
    if (fp != NULL) {
        for (int i = 0; i < numChars; i++) {
            fprintf(fp, "%c", res[i] + 96);
        }
        fprintf(fp, "\n");
        fclose(fp);
    }
}
